#include "hip/hip_runtime.h"
#include "bellman_ford.h"

#include <algorithm>
#include <cfloat>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits>
#include <memory>
#include <random>
#include <stdio.h>
using std::vector;
using std::unique_ptr;
using graph::Graph;
using graph::Edge;

#define CHECK_CUDA_ERROR(ans)                                                  \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU error: %s %s %d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

// Global variables
constexpr double kInfinity = std::numeric_limits<double>::infinity();
constexpr size_t kBlockSize = 512;
constexpr size_t kWordSize = 8;
__device__ double gMaxChange = 0.0;
struct deleter {
  void operator()(void *ptr) { hipFree(ptr); }
};

class CudaEvent final {
public:
  CudaEvent() { hipEventCreate(&event); }
  ~CudaEvent() { hipEventDestroy(event); }
  void wait() { hipEventSynchronize(event); }
  void record() { hipEventRecord(event); }
  float since(const CudaEvent &earlier) {
    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, earlier.event, event);
    return elapsed;
  }

private:
  hipEvent_t event;
};

/**
 * Thin RAII wrapper managing device view and memory of a 1D vector.
 */
template <typename T> struct CudaVector final {
  size_t num_elems;
  size_t num_bytes;
  unique_ptr<T[], deleter> device_ptr;

  CudaVector(size_t num_elems_)
      : num_elems(num_elems_), num_bytes(), device_ptr(nullptr) {
    // Align all allocations to 8-byte boundary
    num_bytes = num_elems_ * sizeof(T);
    size_t rem = num_bytes % kWordSize;
    num_bytes += kWordSize - rem;

    // Malloc memory and zero-initialize it
    CHECK_CUDA_ERROR(hipMalloc((void **)&device_ptr, num_bytes));
    CHECK_CUDA_ERROR(hipMemset(device_ptr.get(), 0, num_bytes));
  }

  CudaVector(const vector<T> &data_)
      : num_elems(data_.size()), num_bytes(0), device_ptr(nullptr) {
    num_bytes = data_.size() * sizeof(T);
    size_t rem = num_bytes % kWordSize;
    num_bytes += kWordSize - rem;

    const char* host_ptr = reinterpret_cast<const char*>(data_.data());

    CHECK_CUDA_ERROR(hipMalloc((void **)&device_ptr, num_bytes));
    CHECK_CUDA_ERROR(hipMemcpy(device_ptr.get(), host_ptr, num_elems * sizeof(T),
          hipMemcpyHostToDevice));
  }

  void copyFromDevice(T *host_ptr) {
    CHECK_CUDA_ERROR(hipMemcpy(host_ptr, device_ptr.get(), num_elems * sizeof(T),
          hipMemcpyDeviceToHost));
  }

  const T *data() const { return device_ptr.get(); }
  T *data() { return device_ptr.get(); }
  void clear() { CHECK_CUDA_ERROR(hipMemset(device_ptr.get(), 0, num_bytes)); }
};

/**
 * Thin RAII wrapper managing device view and memory of a 2D array.
 */
template <typename T> struct CudaArray final {
  size_t num_rows;
  size_t num_cols;
  size_t pitch;
  size_t num_bytes;
  unique_ptr<T[], deleter> device_ptr;

  CudaArray(size_t num_rows_, size_t num_cols_)
      : num_rows(num_rows_), num_cols(num_cols_), pitch(), num_bytes(),
        device_ptr(nullptr) {
    // Allocate 2D pitched memory on the device and get device ptr
    CHECK_CUDA_ERROR(hipMallocPitch((void **)&device_ptr, &pitch,
                                     num_cols * sizeof(T), num_rows));
    CHECK_CUDA_ERROR(
        hipMemset2D(device_ptr.get(), pitch, 0, num_cols, num_rows));
    num_bytes = num_rows * pitch;
  }

  CudaArray(const vector<T> &data_, size_t num_cols_)
      : CudaArray(data_.size() / num_cols_, num_cols_) {
    // Additionally memcpy host src vector to device
    auto host_ptr = reinterpret_cast<const char *>(data_.data());
    CHECK_CUDA_ERROR(hipMemcpy2D(device_ptr.get(), pitch, host_ptr,
                                  num_cols * sizeof(T), num_cols * sizeof(T),
                                  num_rows, hipMemcpyHostToDevice));
  }

  void copyFromDevice(T *host_ptr) {
    CHECK_CUDA_ERROR(hipMemcpy2D(host_ptr, num_cols * sizeof(T),
                                  device_ptr.get(), pitch, num_cols * sizeof(T),
                                  num_rows, hipMemcpyDeviceToHost));
  }

  const T *data() const { return device_ptr.get(); }
  T *data() { return device_ptr.get(); }
  void clear() {
    CHECK_CUDA_ERROR(
        hipMemset2D(device_ptr.get(), pitch, 0, num_cols, num_rows));
  }
};

__global__ void init(unsigned int seed, hiprandState_t *states) {
  hiprand_init(seed, threadIdx.x, 0, &states[threadIdx.x]);
}

// Helper to allow atomicMax to be invoked on a double
// https://github.com/treecode/Bonsai/blob/master/runtime/profiling/derived_atomic_functions.h
__device__ __forceinline__ double atomicMax(double *address, double val) {
  unsigned long long ret = __double_as_longlong(*address);
  while (val > __longlong_as_double(ret)) {
    unsigned long long old = ret;
    if ((ret = atomicCAS((unsigned long long *)address, old,
                         __double_as_longlong(val))) == old)
      break;
  }
  return __longlong_as_double(ret);
}

// Helper to allow atomicMin to be invoked on a double
// https://github.com/treecode/Bonsai/blob/master/runtime/profiling/derived_atomic_functions.h
__device__ __forceinline__ double atomicMin(double *address, double val) {
  unsigned long long ret = __double_as_longlong(*address);
  while (val < __longlong_as_double(ret)) {
    unsigned long long old = ret;
    if ((ret = atomicCAS((unsigned long long *)address, old,
                         __double_as_longlong(val))) == old)
      break;
  }
  return __longlong_as_double(ret);
}

// Templated helper to index into a flat array
template <typename T>
__device__ __forceinline__ T *address(T *src, size_t row, size_t col,
                                      size_t pitch) {
  return (T *)((char *)src + row * pitch) + col;
}

// Helper function for L2 distance
__device__ __forceinline__ double
dist(const double *__restrict__ p0, const double *__restrict__ p1, size_t dim) {
  double distance = 0.0;
  for (int i = 0; i < dim; i++) {
    distance += (p0[i] - p1[i]) * (p0[i] - p1[i]);
  }
  return distance;
}

__device__ __forceinline__ void atomicVecAdd(double *__restrict__ dest,
                                             const double *__restrict__ src,
                                             size_t dim) {
  for (int i = 0; i < dim; i++) {
    atomicAdd(&dest[i], src[i]);
  }
}

__global__ void relax(const size_t num_edges, const double* distsRead, 
    double* distsWrite, const Edge* edges) {

  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index > num_edges) {
    return;
  }

  Edge edge = edges[index];

  double val = distsRead[edge.src] + edge.cost;
  atomicMin(&distsWrite[edge.dest], val);
}

__global__ void copyBack(const size_t num_points, double* distsRead,
    const double* distsWrite) {
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index > num_points) {
    return;
  }

  distsRead[index] = distsWrite[index];
};

BellmanFordOutput bellmanFordCUDA(const Graph &graph, size_t source) {
  auto start = CudaEvent{};
  auto end = CudaEvent{};

  start.record();

  vector<double> localDistances = vector<double>(graph.num_vertices, kInfinity);
  localDistances[source] = 0.0;

  // data structure initialization
  CudaVector<double> distsRead = CudaVector<double>(localDistances);
  CudaVector<double> distsWrite = CudaVector<double>(localDistances);
  CudaVector<Edge> edges = CudaVector<Edge>(graph.getAllEdges());

  // grid and block size calculation
  auto threads = dim3{kBlockSize};
  auto blocks = dim3{};
  blocks.x = (graph.getNumEdges() + threads.x - 1) / threads.x;

  // iteration
  for(size_t iter = 0; iter < graph.num_vertices; iter++) {
    relax<<<blocks, threads>>>(graph.getNumEdges(), distsRead.data(), distsWrite.data(), edges.data());
    copyBack<<<blocks, threads>>>(graph.num_vertices, distsRead.data(), distsWrite.data());
  }

  end.record();
  distsRead.copyFromDevice(localDistances.data());
  end.wait();

  return {localDistances, end.since(start)};
}
