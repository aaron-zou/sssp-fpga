#include "hip/hip_runtime.h"
#include "kmeans.h"

#include <algorithm>
#include <cfloat>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <limits>
#include <memory>
#include <random>
#include <stdio.h>
using std::vector;
using std::unique_ptr;

#define CHECK_CUDA_ERROR(ans)                                                  \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU error: %s %s %d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

// Global variables
constexpr size_t kBlockSize = 512;
constexpr size_t kWordSize = 8;
__device__ double gMaxChange = 0.0;
constexpr typeof(gMaxChange) kZero = 0.0;
struct deleter {
  void operator()(void *ptr) { hipFree(ptr); }
};

class CudaEvent final {
public:
  CudaEvent() { hipEventCreate(&event); }
  ~CudaEvent() { hipEventDestroy(event); }
  void wait() { hipEventSynchronize(event); }
  void record() { hipEventRecord(event); }
  float since(const CudaEvent &earlier) {
    float elapsed = 0.0f;
    hipEventElapsedTime(&elapsed, earlier.event, event);
    return elapsed;
  }

private:
  hipEvent_t event;
};

/**
 * Thin RAII wrapper managing device view and memory of a 1D vector.
 */
template <typename T> struct CudaVector final {
  size_t num_elems;
  size_t num_bytes;
  unique_ptr<T[], deleter> device_ptr;

  CudaVector(size_t num_elems_)
      : num_elems(num_elems_), num_bytes(), device_ptr(nullptr) {
    // Align all allocations to 8-byte boundary
    num_bytes = num_elems_ * sizeof(T);
    size_t rem = num_bytes % kWordSize;
    num_bytes += kWordSize - rem;

    // Malloc memory and zero-initialize it
    CHECK_CUDA_ERROR(hipMalloc((void **)&device_ptr, num_bytes));
    CHECK_CUDA_ERROR(hipMemset(device_ptr.get(), 0, num_bytes));
  }

  const T *data() const { return device_ptr.get(); }
  T *data() { return device_ptr.get(); }
  void clear() { CHECK_CUDA_ERROR(hipMemset(device_ptr.get(), 0, num_bytes)); }
};

/**
 * Thin RAII wrapper managing device view and memory of a 2D array.
 */
template <typename T> struct CudaArray final {
  size_t num_rows;
  size_t num_cols;
  size_t pitch;
  size_t num_bytes;
  unique_ptr<T[], deleter> device_ptr;

  CudaArray(size_t num_rows_, size_t num_cols_)
      : num_rows(num_rows_), num_cols(num_cols_), pitch(), num_bytes(),
        device_ptr(nullptr) {
    // Allocate 2D pitched memory on the device and get device ptr
    CHECK_CUDA_ERROR(hipMallocPitch((void **)&device_ptr, &pitch,
                                     num_cols * sizeof(T), num_rows));
    CHECK_CUDA_ERROR(
        hipMemset2D(device_ptr.get(), pitch, 0, num_cols, num_rows));
    num_bytes = num_rows * pitch;
  }

  CudaArray(const vector<T> &data_, size_t num_cols_)
      : CudaArray(data_.size() / num_cols_, num_cols_) {
    // Additionally memcpy host src vector to device
    auto host_ptr = reinterpret_cast<const char *>(data_.data());
    CHECK_CUDA_ERROR(hipMemcpy2D(device_ptr.get(), pitch, host_ptr,
                                  num_cols * sizeof(T), num_cols * sizeof(T),
                                  num_rows, hipMemcpyHostToDevice));
  }

  void copyFromDevice(T *host_ptr) {
    CHECK_CUDA_ERROR(hipMemcpy2D(host_ptr, num_cols * sizeof(T),
                                  device_ptr.get(), pitch, num_cols * sizeof(T),
                                  num_rows, hipMemcpyDeviceToHost));
  }

  const T *data() const { return device_ptr.get(); }
  T *data() { return device_ptr.get(); }
  void clear() {
    CHECK_CUDA_ERROR(
        hipMemset2D(device_ptr.get(), pitch, 0, num_cols, num_rows));
  }
};

__global__ void init(unsigned int seed, hiprandState_t *states) {
  hiprand_init(seed, threadIdx.x, 0, &states[threadIdx.x]);
}

// Helper to allow atomicMax to be invoked on a double
// https://github.com/treecode/Bonsai/blob/master/runtime/profiling/derived_atomic_functions.h
__device__ __forceinline__ double atomicMax(double *address, double val) {
  unsigned long long ret = __double_as_longlong(*address);
  while (val > __longlong_as_double(ret)) {
    unsigned long long old = ret;
    if ((ret = atomicCAS((unsigned long long *)address, old,
                         __double_as_longlong(val))) == old)
      break;
  }
  return __longlong_as_double(ret);
}

// Templated helper to index into a flat array
template <typename T>
__device__ __forceinline__ T *address(T *src, size_t row, size_t col,
                                      size_t pitch) {
  return (T *)((char *)src + row * pitch) + col;
}

// Helper function for L2 distance
__device__ __forceinline__ double
dist(const double *__restrict__ p0, const double *__restrict__ p1, size_t dim) {
  double distance = 0.0;
  for (int i = 0; i < dim; i++) {
    distance += (p0[i] - p1[i]) * (p0[i] - p1[i]);
  }
  return distance;
}

__device__ __forceinline__ void atomicVecAdd(double *__restrict__ dest,
                                             const double *__restrict__ src,
                                             size_t dim) {
  for (int i = 0; i < dim; i++) {
    atomicAdd(&dest[i], src[i]);
  }
}

// Device function that determines the nearest centroid for each data point
__global__ void assign_centroids(double *points, size_t num_points, size_t dim,
                                 size_t points_pitch, double *old_centroids,
                                 double *centroids, size_t num_centroids,
                                 size_t centroids_pitch, int *counts) {
  // Calculate which point this thread represents
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= num_points)
    return;
  auto point = address<const double>(points, index, 0, points_pitch);

  // Iterate through each centroid to find closest to this point
  double best_distance = DBL_MAX;
  size_t best_index = 0;
  for (int i = 0; i < num_centroids; i++) {
    auto centroid = address<const double>(old_centroids, i, 0, centroids_pitch);
    auto distance = dist(point, centroid, dim);
    if (distance < best_distance) {
      best_distance = distance;
      best_index = i;
    }
  }

  // Atomically update counts and new centroids
  auto centroid = address<double>(centroids, best_index, 0, centroids_pitch);
  atomicVecAdd(centroid, point, dim);
  atomicAdd(&counts[best_index], 1);
}

// Device function that determines the nearest centroid for each data point
__global__ void assign_centroids_private(const double *points, size_t num_points,
                                         size_t dim, size_t points_pitch,
                                         const double *old_centroids,
                                         double *centroids,
                                         size_t num_centroids,
                                         size_t centroids_pitch, int *counts) {
  // Dynamic block-specific shared memory
  extern __shared__ char s[];
  auto local_centroids = reinterpret_cast<double *>(s);
  auto local_counts =
      reinterpret_cast<int *>(s + (num_centroids * centroids_pitch));

  // Calculate which point this thread represents
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= num_points)
    return;
  auto point = address<const double>(points, index, 0, points_pitch);

  // Clear shared memory block
  if (threadIdx.x < num_centroids) {
    local_counts[threadIdx.x] = 0;
    auto local_centroid =
        address<double>(local_centroids, threadIdx.x, 0, centroids_pitch);
    memset(local_centroid, 0, centroids_pitch);
  }
  __syncthreads();

  // Iterate through each centroid to find closest to this point
  double best_distance = DBL_MAX;
  size_t best_index = 0;
  for (int i = 0; i < num_centroids; i++) {
    auto centroid = address<const double>(old_centroids, i, 0, centroids_pitch);
    auto distance = dist(point, centroid, dim);
    if (distance < best_distance) {
      best_distance = distance;
      best_index = i;
    }
  }

  // Atomically update shared memory structures
  auto local_centroid =
      address<double>(local_centroids, best_index, 0, centroids_pitch);
  atomicVecAdd(local_centroid, point, dim);
  atomicAdd(&local_counts[best_index], 1);
  __syncthreads();

  // Aggregate updates to global memory
  if (threadIdx.x < num_centroids) {
    auto dest = address<double>(centroids, threadIdx.x, 0, centroids_pitch);
    auto src =
        address<double>(local_centroids, threadIdx.x, 0, centroids_pitch);
    atomicVecAdd(dest, src, dim);
    atomicAdd(&counts[threadIdx.x], local_counts[threadIdx.x]);
  }
}

// Device function that determines the nearest centroid for each data point
__global__ void
assign_centroids_extra(const double *__restrict__ points, size_t num_points,
                       size_t dim, size_t points_pitch,
                       const double *__restrict__ old_centroids,
                       double *__restrict__ centroids, size_t num_centroids,
                       size_t centroids_pitch, int *__restrict__ counts) {
  // Dynamic block-specific shared memory
  extern __shared__ char s[];
  auto local_centroids = reinterpret_cast<double *>(s);
  auto local_counts =
      reinterpret_cast<int *>(s + (num_centroids * centroids_pitch));

  // Calculate which point this thread represents
  const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= num_points)
    return;
  auto point = address<const double>(points, index, 0, points_pitch);

  // Clear shared memory block
  if (threadIdx.x < num_centroids) {
    local_counts[threadIdx.x] = 0;
    auto local_centroid =
        address<double>(local_centroids, threadIdx.x, 0, centroids_pitch);
    memset(local_centroid, 0, centroids_pitch);
  }
  __syncthreads();

  // Iterate through each centroid to find closest to this point
  double best_distance = DBL_MAX;
  size_t best_index = 0;
  for (int i = 0; i < num_centroids; i++) {
    auto centroid = address<const double>(old_centroids, i, 0, centroids_pitch);
    auto distance = dist(point, centroid, dim);
    if (distance < best_distance) {
      best_distance = distance;
      best_index = i;
    }
  }

  // Atomically update shared memory structures
  auto local_centroid =
      address<double>(local_centroids, best_index, 0, centroids_pitch);
  atomicVecAdd(local_centroid, point, dim);
  atomicAdd(&local_counts[best_index], 1);
  __syncthreads();

  // Aggregate updates to global memory
  if (threadIdx.x < num_centroids) {
    auto dest = address<double>(centroids, threadIdx.x, 0, centroids_pitch);
    auto src =
        address<double>(local_centroids, threadIdx.x, 0, centroids_pitch);
    atomicVecAdd(dest, src, dim);
    atomicAdd(&counts[threadIdx.x], local_counts[threadIdx.x]);
  }
}

// Recalculate the centroids based on which points map to which centroids
__global__ void normalize_centroids(double *points, size_t num_points,
                                    size_t dim, size_t points_pitch,
                                    double *old_centroids, double *centroids,
                                    size_t num_centroids,
                                    size_t centroids_pitch, int *counts,
                                    hiprandState_t *states) {
  // Calculate which centroid this thread represents
  const size_t index = threadIdx.x;
  auto centroid = address<double>(centroids, index, 0, centroids_pitch);

  // Divide each sum to get the new mean
  if (counts[index] > 0) {
    for (size_t i = 0; i < dim; i++) {
      centroid[i] /= counts[index];
    }
  } else {
    // Set random point
    size_t point_index = hiprand(&states[threadIdx.x]) % num_points;
    auto point = address<const double>(points, point_index, 0, points_pitch);
    memcpy(centroid, point, dim * sizeof(double));
  }

  // Update max change for this iteration
  auto old_centroid = address<double>(old_centroids, index, 0, centroids_pitch);
  auto change = sqrt(dist(centroid, old_centroid, dim));
  atomicMax(&gMaxChange, change);

  // Clear the old_centroids (swapped with centroids at end of iter)
  memset(old_centroid, 0, centroids_pitch);
  counts[index] = 0;
}

// Recalculate the centroids based on which points map to which centroids
__global__ void normalize_centroids_extra(
    const double *__restrict__ points, size_t num_points, size_t dim,
    size_t points_pitch, double *__restrict__ old_centroids,
    double *__restrict__ centroids, size_t num_centroids,
    size_t centroids_pitch, int *__restrict__counts, hiprandState_t *states) {
  // Calculate which centroid this thread represents
  const size_t index = threadIdx.x;
  auto centroid = address<double>(centroids, index, 0, centroids_pitch);

  // Divide each sum to get the new mean
  if (counts[index] > 0) {
    for (size_t i = 0; i < dim; i++) {
      centroid[i] /= counts[index];
    }
  } else {
    // Set random point
    size_t point_index = hiprand(&states[threadIdx.x]) % num_points;
    auto point = address<const double>(points, point_index, 0, points_pitch);
    memcpy(centroid, point, dim * sizeof(double));
  }

  // Update max change for this iteration
  auto old_centroid = address<double>(old_centroids, index, 0, centroids_pitch);
  auto change = sqrt(dist(centroid, old_centroid, dim));
  atomicMax(&gMaxChange, change);

  // Clear the old_centroids (swapped with centroids at end of iter)
  memset(old_centroid, 0, centroids_pitch);
  counts[index] = 0;
}

KmeansOutput cuda_kmeans(const vector<double> &data, size_t dim,
                         size_t clusters, size_t iterations, double threshold,
                         bool privatize, bool extra) {
  auto start = CudaEvent{};
  auto end = CudaEvent{};

  // Data structure initialization
  auto dataset = CudaArray<double>(data, dim);
  start.record();
  auto num_points = dataset.num_rows;
  auto counts = CudaVector<int>(clusters);

  // Random initial centroids
  auto init_clusters = vector<double>(clusters * dim);
  std::generate(init_clusters.begin(), init_clusters.end(),
                []() { return static_cast<double>(std::rand()) / RAND_MAX; });
  auto centroids = CudaArray<double>(init_clusters, dim);
  auto old_centroids = CudaArray<double>(clusters, dim);

  // Random initialization for GPU
  auto states = CudaVector<hiprandState_t>(clusters);
  init<<<1, clusters>>>(std::time(0), states.data());

  // Calculate grid and block sizes to guarantee at least one thread per point
  auto threads = dim3{kBlockSize};
  auto blocks = dim3{};
  blocks.x = (num_points + threads.x - 1) / threads.x;

  // Kernel-based k-means implementation
  auto current_max_change = typeof(gMaxChange){};
  auto shared_bytes = centroids.num_bytes + counts.num_bytes;
  size_t iter = 0;
  do {
    std::swap(old_centroids, centroids);

    // Stage where we find nearest centroids and calculate sum
    if (privatize) {
      assign_centroids_private<<<blocks, threads, shared_bytes>>>(
          dataset.data(), num_points, dim, dataset.pitch, old_centroids.data(),
          centroids.data(), clusters, centroids.pitch, counts.data());
    } else if (extra) {
      assign_centroids_extra<<<blocks, threads, shared_bytes>>>(
          dataset.data(), num_points, dim, dataset.pitch, old_centroids.data(),
          centroids.data(), clusters, centroids.pitch, counts.data());
    } else {
      assign_centroids<<<blocks, threads>>>(
          dataset.data(), num_points, dim, dataset.pitch, old_centroids.data(),
          centroids.data(), clusters, centroids.pitch, counts.data());
    }
    CHECK_CUDA_ERROR(hipPeekAtLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Stage where we divide out the sums to get the means and check convergence
    if (privatize || extra) {
      normalize_centroids_extra<<<1, clusters>>>(
          dataset.data(), num_points, dim, dataset.pitch, old_centroids.data(),
          centroids.data(), clusters, centroids.pitch, counts.data(),
          states.data());
    } else {
      normalize_centroids<<<1, clusters>>>(
          dataset.data(), num_points, dim, dataset.pitch, old_centroids.data(),
          centroids.data(), clusters, centroids.pitch, counts.data(),
          states.data());
    }
    CHECK_CUDA_ERROR(hipPeekAtLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Check convergence and swap old_centroids and centroids
    CHECK_CUDA_ERROR(hipMemcpyFromSymbol(&current_max_change, HIP_SYMBOL(gMaxChange),
                                          sizeof(current_max_change)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gMaxChange), &kZero, sizeof(kZero)));
  } while ((++iter < iterations || iterations == 0) &&
           (current_max_change > threshold));

  // Copy back final centroids data
  end.record();
  auto host_centroids = vector<double>(clusters * dim * sizeof(double));
  centroids.copyFromDevice(host_centroids.data());
  end.wait();

  return {iter, host_centroids, end.since(start)};
}
